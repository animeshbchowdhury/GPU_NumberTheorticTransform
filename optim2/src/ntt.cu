#include "hip/hip_runtime.h"
#include <cmath>		/* log2(), pow() */
#include <cstdint>		/* uint64_t */
#include <cstdlib> 		/* malloc() */
#include <iostream>

#include "../include/utils.cuh"	/* bit_reverse(), modExp(), modulo() */
#include "../include/ntt.h" 	//INCLUDE HEADER FILE


__global__ void blockComputation(uint64_t*,uint64_t*,uint64_t,uint64_t,uint64_t,uint64_t*,uint64_t,uint64_t) ;
void blockComp(uint64_t* , uint64_t ,uint64_t,uint64_t,uint64_t*,uint64_t) ;

void cpuToGpuMemcpy(void* h_data,void* d_data,int size)
{
    hipError_t err = hipMemcpy(d_data,h_data,size,hipMemcpyHostToDevice) ;
    if(err != hipSuccess)
    {
            fprintf(stderr,"Failed to copy vector from host device!",hipGetErrorString(err)) ;
            exit(EXIT_FAILURE) ;
    }
}

void gpuToCpuMemcpy(uint64_t* d_data,uint64_t* h_data,int size)
{
    hipError_t err = hipMemcpy(h_data,d_data,size,hipMemcpyDeviceToHost) ;
    if(err != hipSuccess)
    {
            fprintf(stderr,"Failed to copy vector from gpu device!",hipGetErrorString(err)) ;
            exit(EXIT_FAILURE) ;
    }
    hipFree(d_data) ;
}


/**
 * Perform an in-place iterative breadth-first decimation-in-time Cooley-Tukey NTT on an input vector and return the result
 *
 * @param vec 	The input vector to be transformed
 * @param n	The size of the input vector
 * @param p	The prime to be used as the modulus of the transformation
 * @param r	The primitive root of the prime
 * @param rev	Whether to perform bit reversal on the input vector
 * @return 	The transformed vector
 */



uint64_t *inPlaceNTT_DIT(uint64_t *vec, uint64_t n, uint64_t p, uint64_t r, uint64_t* twiddleFactors, bool rev){

	uint64_t *result,*result_cpu;

	uint64_t m, k_, a ;
        uint64_t factor1, factor2 ;
	result = (uint64_t *) malloc(n*sizeof(uint64_t));
	result_cpu = (uint64_t *) malloc(n*sizeof(uint64_t));

	if(rev){
		result = bit_reverse(vec, n);
		result_cpu = bit_reverse(vec, n);
	}else{
		for(uint64_t i = 0; i < n; i++){	
			result[i] = vec[i];
			result_cpu[i] = vec[i];
		}
	}

	for(uint64_t i = 1; i <= log2(n); i++){ 

		m = pow(2,i);
		k_ = (p - 1)/m;
		a = modExp(r,k_,p);
        
		for(uint64_t j = 0; j < n; j+=m){

			for(uint64_t k = 0; k < m/2; k++){

				factor1 = result_cpu[j + k];
				factor2 = modulo(modExp(a,k,p)*result_cpu[j + k + m/2],p);
			
				result_cpu[j + k] 	= modulo(factor1 + factor2, p);
				result_cpu[j + k+m/2] 	= modulo(factor1 - factor2, p);
			}
		}
                blockComp(result,n,m,p,twiddleFactors,i-1) ;

	}
	bool compCPUGPUResult = compVec(result,result_cpu,n,true) ;
	std::cout<<"\nComparing output of cpu and gpu :"<<compCPUGPUResult ;
	return result;

}

void blockComp(uint64_t* res, uint64_t resLength,uint64_t blockSize,uint64_t p,uint64_t* twiddleFactors,uint64_t rowInfoProcessing)
{
    uint64_t *cuda_result, *cuda_output  ;
    uint64_t sizeOfRes = resLength*sizeof(uint64_t) ;
    uint64_t *preComputeTwiddle ;
    uint64_t rowTwiddle= log2(resLength) ; 
    uint64_t columnTwiddle = resLength/2 ;
    hipMalloc(&cuda_result,sizeOfRes) ;
    hipMalloc(&cuda_output,sizeOfRes) ;
    hipMalloc(&preComputeTwiddle,rowTwiddle*columnTwiddle*sizeof(uint64_t)) ;
    cpuToGpuMemcpy(res,cuda_result,sizeOfRes) ;
    cpuToGpuMemcpy(twiddleFactors,preComputeTwiddle,rowTwiddle*columnTwiddle*sizeof(uint64_t)) ;

    int tpb = 32;//blockSize;
    int bpg = (resLength -1 + tpb)/tpb ;

    
    
    blockComputation<<<bpg,tpb>>>(cuda_result,cuda_output,resLength,blockSize,p,preComputeTwiddle,columnTwiddle,rowInfoProcessing) ;
    hipError_t err = hipGetLastError() ;

	if(err != hipSuccess)
	{
	    fprintf(stderr,"Issues in running the kernel",hipGetErrorString(err)) ;
            exit(EXIT_FAILURE) ;
	}

    gpuToCpuMemcpy(cuda_output,res,sizeOfRes) ;
    hipFree(cuda_result) ;
    hipFree(preComputeTwiddle) ;
}

__global__ void blockComputation(uint64_t* result, uint64_t* output,uint64_t n,uint64_t m,uint64_t p,uint64_t* twiddleFactors,uint64_t maxTwiddleCols,uint64_t rowInfoProcessing)
{
    uint64_t idx=blockDim.x*blockIdx.x+threadIdx.x ;
    uint64_t k ;
    uint64_t factor1,factor2;
    if(idx < n)
	{
	k = idx%m ;
	if(k < m/2)
	{
		factor1 = result[idx] ;
		factor2 = modulo(twiddleFactors[rowInfoProcessing*maxTwiddleCols + k]*result[idx+m/2],p);	
		output[idx] = modulo(factor1+factor2,p) ;
	}
	else
	{
		factor1 = result[idx - m/2] ;
		factor2 = modulo(twiddleFactors[rowInfoProcessing*maxTwiddleCols + k-(m/2)]*result[idx],p) ;
		output[idx] = modulo(factor1-factor2,p) ;
	}
    }
}
